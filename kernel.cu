#include "hip/hip_runtime.h"
/**
* The Binary Classification program implements the Binary Classification algorithm in parallel development,
* using MPI,OpenMP and Cuda (By Invidia)
*
* @author Sagiv Asraf
* @Id : 312527450
* @since 25-08-2019
* Lecturer : Dr. Boris Moroz.
*
*/

#include "MainApp.h"

#define NUM_OF_THREADS_PER_BLOCK 1024

__global__ void calcSign(int numOfPoints, int *expectedSigns, double* pointsArray, double* weightsArray, int weightsVectorSize)
{

	/*CudaThread -> as index of point to check. 
	For example: 
	CudaThread: 20032 check the point 20032 from the points array and so on.
	*/

	int cudaThread = threadIdx.x + (blockIdx.x * blockDim.x);

	if (cudaThread >= numOfPoints) {
		return;
	}

	double sum = 0;

	for (int i = 0; i < weightsVectorSize; i++)
	{
		/*	Each point has a weight, we use the next formula for calculate the sign of the mulpilicity
		between the weights and the point's coordiantes.	*/
		sum += weightsArray[i] * pointsArray[(cudaThread * weightsVectorSize) + i];
	}

	sum >= 0 ? expectedSigns[cudaThread] = 1 : expectedSigns[cudaThread] = -1;

}

// Helper method for using CUDA to calcuate the sign of the points via the weights vector.
hipError_t signCalculationWithCuda(int* expectedSignsArray, double* points, int numOfPoints, double* weightsVector, int weightsVectorSize)
{

	double* pointsArray_Cuda = 0;
	double* weightsArray_Cuda = 0;
	int* expctedArray_Cuda = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("\n***hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n***");
		goto Error;
	}

	/*Allocate memory for pointsArray_Cuda*/
	cudaStatus = hipMalloc((void**)&pointsArray_Cuda, sizeof(double) * (numOfPoints * weightsVectorSize));
	if (cudaStatus != hipSuccess) {
		printf("\n***hipMalloc failed!\n***");
		goto Error;
	}

	//Copy input arrays from host memory to GPU buffers.
	cudaStatus = hipMemcpy(pointsArray_Cuda, points, sizeof(double)  * (numOfPoints * weightsVectorSize), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("\n***hipMemcpy Of PointsArray failed!\n***");
		goto Error;
	}
	/*Allocate memory for weightsArray_Cuda*/
	cudaStatus = hipMalloc((void**)&weightsArray_Cuda, weightsVectorSize * sizeof(double));
	if (cudaStatus != hipSuccess) {
		printf("\n***hipMalloc failed!\n***");
		goto Error;
	}

	/*Allocate memory for expctedArray_Cuda*/
	cudaStatus = hipMalloc((void**)&expctedArray_Cuda, numOfPoints * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("\n***hipMalloc failed!\n***");
		goto Error;
	}

	//Copy input arrays from host memory to GPU buffers.
	cudaStatus = hipMemcpy(weightsArray_Cuda, weightsVector, weightsVectorSize * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("\n***hipMemcpy failed!\n***");
		goto Error;
	}

	/* Nvidia formula
	https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
	*/

	int numOfBlocks = (numOfPoints + NUM_OF_THREADS_PER_BLOCK - 1) / NUM_OF_THREADS_PER_BLOCK;

	// Launch a kernel on the GPU with numOfBlocks blocks and NUM_OF_THREADS_PER_BLOCK threads per each block.
	calcSign << <numOfBlocks, NUM_OF_THREADS_PER_BLOCK >> >(numOfPoints, expctedArray_Cuda, pointsArray_Cuda, weightsArray_Cuda, weightsVectorSize);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("\n***calcSign launch failed: %s\n\n***", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Copy output array from GPU buffer to host memory.
	cudaStatus = hipMemcpy(expectedSignsArray, expctedArray_Cuda, (numOfPoints * sizeof(int)), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("\n***hipMemcpy LAST failed!\n\n***");
		goto Error;
	}

Error:
	hipFree(expctedArray_Cuda);
	hipFree(pointsArray_Cuda);
	hipFree(weightsArray_Cuda);

	return cudaStatus;
}